
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <ctime>

#define MAX_RANDOM 2147483647 
#define NMAX 100000

#define DEBUG     1   //set level of debug visibility [0=>off,1=>min,2=>max]
#define NOISEOFF  0   //set to suppress noise in channel

#define N_ITERATION 2 //no. of turbo decoder iterations
// #define TBD -1        //trellis termination bits (inserted by encoder #1)

//  Block length in our example (6 databits + 2 termination bits)
//  A practical turbo code must use a much larger block length to
//  approach the Shannon limit.
//
#define N 16000
#define permutationseed 3

//  Pseudo random permutation (of length = the block length).
//  In our example we use a PN sequence with 0 appended
//  See chapter 3 in [1] for some other possibilities.
//



// tables for trellis are global (simple example from pp 38 in [1])
//
#define M  4      //no. of trellis states
int X[N];
int permutation[N];

int from[M][2];   //from[m][i] = next state (from state m with databit = i)
int to[M][2];     //to[m][i] = previous state (to state m with databit = i)
int parity[M][2]; //parity bit associated with transition from state m
int term[M][2];   //term[m] = pair of data bits required to terminate trellis


void randomInterleaver(){

int interleaver[NMAX];
int check[NMAX];                       // Already permuted positions
int i;
int position;


  srandom(permutationseed);

  for (i=0; i<N; i++)
    check[i] = 0;

  for (i=0; (i<N); i++)
    {
      do
        position = (int) ( ( (double)(random())/MAX_RANDOM ) * N );
      while ( check[position] );
  

       check[position] = 1;			
      interleaver[i] = position;
    }    

  for (i=0; i<N; i++)
    {
	permutation[i]=interleaver[i];
	X[i]=interleaver[i]%2;
//       printf("%5d -> %5d\n",X[i],permutation[i]);
    	
    }


}


//  Normally distributed number generator (ubiquitous Box-Muller method)
//
double normal(void)
{
	double x, y, rr, randn;
	do{
        x  = (double) 2*rand()/RAND_MAX - 1.0; //uniform in range [-1,1]
        y  = (double) 2*rand()/RAND_MAX - 1.0; //uniform in range [-1,1]
        rr = x*x + y*y;
    } while( rr >= 1 );
    randn = x*sqrt((-2.0*log(rr))/rr);
  return(randn);
}

//  modified BCJR algorithm (MAP decoder)
//
__global__ void calgamma(double *d_gammae,double *d_gamma,int *d_parity,double *d_La,double *d_x_d,double *d_p_d,int Lc)
	{
	int i = blockIdx.x*400+threadIdx.x;
    	int j = blockIdx.y;
	int k = blockIdx.z;	
	double xk_h;
	double pk_h;
	xk_h=k ? +1 : -1;
	pk_h=d_parity[j*2+k] ? +1 : -1;
	d_gamma[M*2*i+2*j+k]=exp(0.5*(d_La[i] * xk_h + Lc * d_x_d[i] * xk_h +
				                          Lc * d_p_d[i] * pk_h));
	d_gammae[M*2*i+2*j+k] = exp(0.5*(Lc * d_p_d[i] * pk_h));

	}


__global__ void calExtLLR(double *d_gammae,double *d_alpha,double *d_beta,int *d_from,double *d_Le)
	{
		int k = blockIdx.x*400+threadIdx.x;
		double pr1,pr0;
		pr1=0;
		pr0=0;
		int m;	

		for(m = 0; m < 4; m++)
		{
			//we use gammae rather than gamma as we want the
			//extrinsic component of the overall likelihood
			pr1 += (d_alpha[k*M+m] * d_gammae[k*M*2+m*2+1] * d_beta[(k+1)*M+d_from[m*2+1]]);
			pr0 += (d_alpha[k*M+m] * d_gammae[k*M*2+m*2+0] * d_beta[(k+1)*M+d_from[m*2+0]]);
		}
		d_Le[k] = log(pr1/ pr0); //extrinsic likelihood
	}

__global__ void calAlpha(double *d_gamma,double *d_alpha,int *d_to){
	__shared__ double *dd_gamma;
	__shared__ int *dd_to;
	dd_gamma = d_gamma;
	dd_to = d_to;
	
	
	__shared__ double total;
	int l=blockIdx.x;
	//printf("\n\n\nl=%d\n\n\n",l);
	int bitsPerBlock=8000;
	int k,m;
	
	d_alpha[(l*bitsPerBlock+0)*4+0] = 1;
	for(m = 1; m < M; m++)
		d_alpha[(l*bitsPerBlock+0)*4+m] = 0;

	for(k = 1; k <= bitsPerBlock; k++)
	{
		total = 0;

	    for(m = 0; m < M; m++)
	    {
		d_alpha[(l*bitsPerBlock+k)*4+m] = d_alpha[(l*bitsPerBlock+(k-1))*4+dd_to[m*2+0]] * dd_gamma[(l*bitsPerBlock+(k-1))*4*2 + dd_to[m*2+0]*2+0] + d_alpha[(l*bitsPerBlock+(k-1))*4+dd_to[m*2+1]] * dd_gamma[(l*bitsPerBlock+(k-1))*4*2+dd_to[m*2+1]*2+1];

			total += d_alpha[(l*bitsPerBlock+k)*4+m];
		}

		//normalise
		for(m = 0; m < M; m++)
			d_alpha[(l*bitsPerBlock+k)*4+m] /= total;
	}

}
__global__ void calBeta(double *d_gamma,double *d_beta,int *d_from,int is_term){
	__shared__ double *dd_gamma;
	__shared__ int *dd_from;
	dd_gamma = d_gamma;
	dd_from = d_from;
	
	
	__shared__ double total;
	int l=blockIdx.x;
	int bitsPerBlock=8000;
	int k,m;
	
	if(is_term)                 //if trellis terminated
	{
		//we know for sure the final state is 0
	    d_beta[((l+1)*bitsPerBlock)*4+0] = 1;
	    for(m = 1; m < M; m++)
	    	d_beta[((l+1)*bitsPerBlock)*4+m] = 0;
	}
	else                       //else trellis not terminated
	{
		//we haven't a clue which is final state
		//so the best we can do is say they're all equally likely
	    for(m = 0; m < M; m++)
	    	d_beta[((l+1)*bitsPerBlock)*4+m] = 1.0 / (double) M;
	}

    //iterate backwards through trellis
	for(k = bitsPerBlock-1; k >= 0; k--)
	{
		total = 0;
		for(m = 0; m < 4; m++)
		{
			d_beta[(l*bitsPerBlock+k)*4+m] = d_beta[(l*bitsPerBlock+(k+1))*4+dd_from[m*2+0]] * dd_gamma[(l*bitsPerBlock+k)*4*2+m*2+0] +
				         d_beta[(l*bitsPerBlock+(k+1))*4+dd_from[m*2+1]] * dd_gamma[(l*bitsPerBlock+k)*4*2+m*2+1];


			total += d_beta[(l*bitsPerBlock+k)*4+m];
		}

        //normalise
		for(m = 0; m < M; m++)
			d_beta[(l*bitsPerBlock+k)*4+m] /= total;
	}


}


void modified_bcjr
(
	int    is_term,      //indicates if trellis terminated
	double Lc,           //Lc = 2/(sigma*sigma) = channel reliability
	double La[N],        //apriori likelihood of each info bit
	double x_d[N],       //noisy data
	double p_d[N],       //noisy parity
	double Le[N]         //extrinsic log likelihood
)
{
	int    k, m, i;
	double xk_h, pk_h;      //databit & parity associated with a branch
	double gamma[N][M][2];  //gammas for total likelihoods
	double gammae[N][M][2]; //gammas for extrinsic likelihoods
	double alpha[N+1][M];   //probability of entering branch via state m
	double beta[N+1][M];    //probability of exiting branch via state m
	double total;           //used for normalising alpha's and beta's

    
    
    //calculate branch gamma's
	double *d_gammae;
	double *d_gamma;
	int *d_parity;
	double *d_La;
	double *d_x_d;
	double *d_p_d;
	

	hipMalloc((void**)&d_gammae,N*M*2*sizeof(double));
	hipMalloc((void**)&d_gamma,N*M*2*sizeof(double));
	hipMalloc((void**)&d_parity,M*2*sizeof(int));
	hipMalloc((void**)&d_La,N*sizeof(double));
	hipMalloc((void**)&d_x_d,N*sizeof(double));
	hipMalloc((void**)&d_p_d,N*sizeof(double));
    
	
	hipMemcpy(d_parity,parity,M*2*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_x_d,x_d,N*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_La,La,N*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_p_d,p_d,N*sizeof(double),hipMemcpyHostToDevice);
    
	dim3 grid(N/400,M,2);

	calgamma<<<grid,400>>>(d_gammae,d_gamma,d_parity,d_La,d_x_d,d_p_d,Lc);

	hipMemcpy(gamma,d_gamma,M*N*2*sizeof(double),hipMemcpyDeviceToHost);
//	cudaMemcpy(gammae,d_gammae,M*N*2*sizeof(double),cudaMemcpyDeviceToHost);
		
	hipFree(d_gamma);
//	cudaFree(d_gammae);
	hipFree(d_parity);
	hipFree(d_La);
	hipFree(d_x_d);
	hipFree(d_p_d);


	



	//  Calculate state alpha's
	//
    //  As the likelihood ratio for each stage k has a linear combination
    //  of alpha terms in both the numerator and the denominator, we can
    //  scale all the alpha's by any convenient scaling constant.
    //
    //  To help avoid underflow/overflow we normalise the alphas at each
    //  stage so that sum across all states is unity.
    //

    //  The encoders always start in state 0

/*	alpha[0][0] = 1;
	for(m = 1; m < M; m++)
		alpha[0][m] = 0;

	for(k = 1; k <= N; k++)
	{
		total = 0;

	    for(m = 0; m < M; m++)
	    {
			alpha[k][m] = alpha[k-1][to[m][0]] * gamma[k-1][to[m][0]][0] +
			              alpha[k-1][to[m][1]] * gamma[k-1][to[m][1]][1];

			total += alpha[k][m];
		}

		//normalise
		for(m = 0; m < M; m++)
			alpha[k][m] /= total;
	}
*/


	double *d_alpha;
	int *d_to;
	
	hipMalloc((void**)&d_gamma,N*M*2*sizeof(double));
	hipMalloc((void**)&d_alpha,(N+1)*M*sizeof(double));
	hipMalloc((void**)&d_to,M*2*sizeof(int));
	
	
	//cudaMemcpy(d_alpha,alpha,(N+1)*M*sizeof(double),cudaMemcpyHostToDevice);
	hipMemcpy(d_to,to,M*2*sizeof(int),hipMemcpyHostToDevice);
    	hipMemcpy(d_gamma,gamma,M*N*2*sizeof(double),hipMemcpyHostToDevice);

	calAlpha<<<N/8000,1>>>(d_gamma,d_alpha,d_to);

	
	hipMemcpy(alpha,d_alpha,(N+1)*M*sizeof(double),hipMemcpyDeviceToHost);

	
	hipFree(d_to);
	hipFree(d_gamma);
	hipFree(d_alpha);


	//  Calculate state beta's
	//
    //  As the likelihood ratio for each stage k has a linear combination
    //  of beta terms in both the numerator and the denominator, we can
    //  scale all the beta's by any convenient scaling constant.
    //
    //  To help avoid underflow/overflow we normalise the betas at each
    //  stage so that sum across all states is unity.
    //


	double *d_beta;
	int *d_from;
	
	hipMalloc((void**)&d_gamma,N*M*2*sizeof(double));
	hipMalloc((void**)&d_beta,(N+1)*M*sizeof(double));
	hipMalloc((void**)&d_from,M*2*sizeof(int));
	
	
	//cudaMemcpy(d_alpha,alpha,(N+1)*M*sizeof(double),cudaMemcpyHostToDevice);
	hipMemcpy(d_from,from,M*2*sizeof(int),hipMemcpyHostToDevice);
    	hipMemcpy(d_gamma,gamma,M*N*2*sizeof(double),hipMemcpyHostToDevice);

	calBeta<<<N/8000,1>>>(d_gamma,d_beta,d_from,is_term);

	
	hipMemcpy(beta,d_beta,(N+1)*M*sizeof(double),hipMemcpyDeviceToHost);

	hipFree(d_beta);
	hipFree(d_from);
	hipFree(d_gamma);


    //  Calculate extrinsic likelihood
    //
	//  This is the information "gleaned" from the parity check
	//  Note the Ck's in equation 20 in [2] are different in the
	//  numerator and denominator. This is why the channel and
	//  apriori likelihoods can be brought out.
	//


//	double *d_alpha;
	//double *d_beta;
	//int *d_from;
	double *d_Le;
	
	
	hipMalloc((void**)&d_alpha,(N+1)*M*sizeof(double));
	hipMalloc((void**)&d_beta,(N+1)*M*sizeof(double));
	hipMalloc((void**)&d_from,M*2*sizeof(int));
	hipMalloc((void**)&d_Le,N*sizeof(double));
	
	
	hipMemcpy(d_alpha,alpha,(N+1)*M*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_beta,beta,(N+1)*M*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_from,from,M*2*sizeof(int),hipMemcpyHostToDevice);
    

	calExtLLR<<<N/400,400>>>(d_gammae,d_alpha,d_beta,d_from,d_Le);

	
	hipMemcpy(Le,d_Le,N*sizeof(double),hipMemcpyDeviceToHost);

	hipFree(d_gammae);
	hipFree(d_alpha);
	hipFree(d_beta);
	hipFree(d_Le);



    #if DEBUG > 1
    for(k = 0; k < N; k++)
    {
		for(m = 0; m < M; m++)
		{
			for(i = 0; i < 2; i++)
			{
				printf("gamma[%i][%i][%i]  = %f\t", k, m, i, gamma[k][m][i]);
				printf("gammae[%i][%i][%i] = %f\n", k, m, i, gammae[k][m][i]);
			}
		}
		printf("\n");
	}

	for(k = 0; k <= N; k++)
	{
	    for(m = 0; m < M; m++)
			printf("alpha[%i][%i] = %f\n", k, m, alpha[k][m]);
		printf("\n");
	}
	for(k = 0; k <= N; k++)
	{
	    for(m = 0; m < M; m++)
			printf("beta[%i][%i] = %f\n", k, m, beta[k][m]);
		printf("\n");
	}
    #endif

}

//
//      +--------------------------> Xk
//      |  fb
//      |  +---------(+)-------+
//      |  |          |        |
//  Xk--+-(+)-+->[D]----->[D]--+
//            |                |
//            +--------------(+)---> Pk
//
//
void gen_tab(void)
{
	int m, i, b0, b1, fb, state;

    //generate tables for 4 state RSC encoder
	for(m = 0; m < M; m++) //for each starting state
		for(i = 0; i < 2; i++) //for each possible databit
		{
			b0 = (m >> 0) & 1; //bit 0 of state
			b1 = (m >> 1) & 1; //bit 1 of state

			//parity from state m with databit i
			parity[m][i] = b0 ^ i;

			//from[m][i] = next state from state m with databit i
			from[m][i]   = (b0<<1) + (i ^ b0 ^ b1);
		}

    //to[m][i] = previous state to state m with databit i
    for(m = 0; m < M; m++)
    	for(i = 0; i < 2; i++)
			to[from[m][i]][i] = m;

	//  Generate table of data bit pairs which terminate
	//  the trellis for a given state m
	//
	//  We simply set Xk equal to the feedback value to force
	//  the delay line to fill up with zeros.
	//
	for(m = 0; m < M; m++) //for each state
	{
		state = m;
		b0 = (state >> 0) & 1; //bit 0 of state
		b1 = (state >> 1) & 1; //bit 1 of state
		fb = b0 ^ b1;          //feedback bit
		term[m][0] = fb;       //will set X[N-2] = fb

		state = from[m][fb];   //advance from state m with databit=fb
		b0 = (state >> 0) & 1; //bit 0 of state
		b1 = (state >> 1) & 1; //bit 1 of state
		fb = b0 ^ b1;          //feedback bit
		term[m][1] = fb;       //will set X[N-1] = fb
	}
}

//
//       +-----------> Xk
//       |
//       |
//       |
//  Xk---+--[E1]-----> P1k
//       |
//      [P]
//       |
//       +--[E2]-----> P2k
//
//
void turbo_encode
(
	int X[N],   //block of N-2 information bits + 2 to_be_decided bits
	int P1[N],  //encoder #1 parity bits
	int P2[N]   //encoder #2 parity bits
)
{
	int    k;      //trellis stage
	int    state;  //encoder state
	int    X_p[N]; //X_permuted = permuted bits

	//encoder #1
	state = 0; //encoder always starts in state 0
	for(k = 0; k < N-2; k++)
	{
		P1[k] = parity[state][X[k]];
		state = from[state][X[k]];
		//printf("s[%i] = %i\n", k, state);
	}

	//terminate encoder #1 trellis to state 0
	X[N-2]  = term[state][0];  //databit to feed a 0 into delay line
	X[N-1]  = term[state][1];  //databit to feed another 0 into delay line

	P1[N-2] = parity[state][X[N-2]]; //parity from state with databitX[N-2]
	state   = from[state][X[N-2]];   //next state from current state
    P1[N-1] = parity[state][X[N-1]]; //parity from state with databit=X[N-1]
	state   = from[state][X[N-1]];   //next state from current state

	if(state != 0)
	{
		//should never get here
		printf("Error: Could not terminate encoder #1 trellis\n");
		exit(1);
	}

	//permute tx databits for encoder #2
	for(k = 0; k < N; k++)
		X_p[k] = X[permutation[k]];

	//encoder #2
	state = 0; //encoder always starts in state 0
	for(k = 0; k < N; k++)
	{
		P2[k] = parity[state][X_p[k]]; //parity from state with databit=X_p[k]
		state = from[state][X_p[k]];   //next state from current state
	}

	//for(k = 0; k < N; k++)
	//	printf("%i %i %i %i\n", X[k], P1[k], X_p[k], P2[k]);

}

void turbo_decode(
	double sigma,   //channel noise standard deviation
	double x_d[N],  //x_dash  = noisy data symbol
	double p1_d[N], //p1_dash = noisy parity#1 symbol
	double p2_d[N], //p2_dash = noisy parity#2 symbol
	double L_h[N],  //L_hat = likelihood of databit given entire observation
	int    X_h[N]   //X_hat = sliced MAP decisions
)
{
	int i, k;

	double Le1[N];    //decoder #1 extrinsic likelihood
	double Le1_p[N];  //decoder #1 extrinsic likelihood permuted
	double Le2[N];    //decoder #2 extrinsic likelihood
	double Le2_ip[N]; //decoder #2 extrinsic likelihood inverse permuted
    double Lc;        //channel reliability value

    Lc = 2.0 / (sigma*sigma); //requires sigma to be non-trivial

    //zero apriori information into very first iteration of BCJR
    for(k = 0; k < N; k++)
		Le2_ip[k] = 0;

    for(i = 0; i < N_ITERATION; i++)
    {
    	modified_bcjr(1, Lc, Le2_ip, x_d, p1_d, Le1);

        //permute decoder#1 likelihoods to match decoder#2 order
    	for(k = 0; k < N; k++)
    		Le1_p[k] = Le1[permutation[k]];

    	modified_bcjr(0, Lc, Le1_p,  x_d, p2_d, Le2);

        //inverse permute decoder#2 likelihoods to match decoder#1 order
    	for(k = 0; k < N; k++)
    		Le2_ip[permutation[k]] = Le2[k];

        #if DEBUG > 1
		for(k = 0; k < N; k++)
		{
 			printf("Le1[%i] = %f\t", k, Le1[k]);
 			printf("Le2_ip[%i] = %f\t", k, Le2_ip[k]);
 			//printf("Lc*x_d[%i] = %f", k, Lc*x_d[k]);
			printf("\n");
		}
		printf("\n");
		#endif
	}

    //calculate overall likelihoods and then slice'em
    for(k = 0; k < N; k++)
    {
		L_h[k] = Lc*x_d[k] + Le1[k] + Le2_ip[k]; //soft decision
		X_h[k] = (L_h[k] > 0.0) ? 1 : 0;         //hard decision
	}
}

/*
gcc turbo_example.c -lm -o t; t
*/

int main(void)
{
   
    	randomInterleaver();
	float snr;
	int snrdb;
	double noise;
	int signal_power=1;	

	printf("\n\n****INPUT****\n\n");

	int    k;         //databit index (trellis stage)

	FILE *fp=fopen("snrber_gpu.dat","a+");
    	//noise standard deviation
	double sigma = 1.0;

	int    P1[N];     //encoder #1 parity bits
	int    P2[N];     //encoder #2 parity bits
	double x[N];      //databit mapped to symbol
	double p1[N];     //encoder #1 parity bit mapped to symbol
	double p2[N];     //encoder #2 parity bit mapped to symbol
	double x_d[N];    //x_dash  = noisy data symbol
	double p1_d[N];   //p1_dash = noisy parity#1 symbol
	double p2_d[N];   //p2_dash = noisy parity#2 symbol
	double L_h[N];    //L_hat = likelihood of databit given entire observation
	int    X_h[N];    //X_hat = sliced MAP decisions
	clock_t t1, t2;
	double time_count;
	double elapsed;
    /********************************
     *         INITIALISE           *
     ********************************/
	printf("Enter the SNR value in db");
	scanf("%d",&snrdb);	
	printf("%d",snrdb);
	snr= pow(10,(float)snrdb/10);
	
	noise = (float)signal_power/snr;

	srand(1);    //init random number generator
  	gen_tab();   //generate trellis tables
	sigma  = sqrt(noise);  //noise std deviation


    /********************************
     *           ENCODER            *
     ********************************/

	turbo_encode(X, P1, P2);

    //map bits to symbols
	for(k = 0; k < N; k++) //for entire block length
	{
		x[k]  = X[k]  ? +1 : -1;  //map databit to symbol
		p1[k] = P1[k] ? +1 : -1;  //map parity #1 to symbol
		p2[k] = P2[k] ? +1 : -1;  //map parity #2 to symbol
	}

    /********************************
     *           CHANNEL            *
     ********************************/

    //add some AWGN
	for(k = 0; k < N; k++)
	{
		#if NOISEOFF
		x_d[k]  = x[k];
		p1_d[k] = p1[k];
		p2_d[k] = p2[k];
		#else
		x_d[k]  = x[k]  + sigma*normal();
		p1_d[k] = p1[k] + sigma*normal();
		p2_d[k] = p2[k] + sigma*normal();
		#endif
	}

    #if DEBUG > 1
	for(k = 0; k < N; k++)
		printf("%f \t%f \t%f\n", x_d[k], p1_d[k], p2_d[k]);
	#endif

    /********************************
     *           DECODER            *
     ********************************/
	t1 = clock();
    	turbo_decode(sigma, x_d, p1_d, p2_d, L_h, X_h);
	t2 = clock();

	int count=0;
	float ber;
	for(k=0; k < N; k++) {
		if(X_h[k] != X[k])
		count++;
	}
	ber=(float)count/N;
	printf("BER is %f",ber);
	printf("count is %d",count);
	fprintf(fp,"%d %f",snrdb, ber);
        fprintf(fp,"\n");
   	fclose(fp);

    	time_count = (float) t2 - (float) t1;
   	elapsed = time_count / CLOCKS_PER_SEC * 1000;
	printf("\n\n Time elapsed =%lf ms\n",elapsed);
    
    
	return 0;

}

